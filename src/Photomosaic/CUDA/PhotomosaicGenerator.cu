/*
	Copyright © 2018-2020, Morgan Grundy

	This file is part of Mosaic Magnifique.

    Mosaic Magnifique is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Mosaic Magnifique is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Mosaic Magnifique.  If not, see <https://www.gnu.org/licenses/>.
*/

#include <hip/hip_math_constants.h>
#include <algorithm>
#include <hip/hip_runtime.h>


#include <math.h>

//Calculates the euclidean difference between main image and library images
__global__
void euclideanDifferenceKernel(float *im_1, float *im_2, size_t noLibIm,
                               unsigned char *mask_im,
                               size_t size, size_t channels, size_t *target_area, double *variants)
{
    const size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * channels;
    const size_t stride = blockDim.x * gridDim.x * channels;
    for (size_t i = index; i < size * size * channels * noLibIm; i += stride)
    {
        const size_t im_1_index = i % (size * size * channels);
        const size_t grayscaleIndex = im_1_index / channels;

        const size_t row = grayscaleIndex / size;
        if (row < target_area[0] || row >= target_area[1])
        {
            variants[i / channels] = 0;
            continue;
        }

        const size_t col = grayscaleIndex % size;
        if (col < target_area[2] || col >= target_area[3])
        {
            variants[i / channels] = 0;
            continue;
        }

        if (mask_im[grayscaleIndex] == 0)
            variants[i / channels] = 0;
        else
            variants[i / channels] = sqrt(pow((double) (im_1[im_1_index] - im_2[i]), (double)2.0) +
                                          pow((double) (im_1[im_1_index + 1] - im_2[i + 1]), (double)2.0) +
                                          pow((double) (im_1[im_1_index + 2] - im_2[i + 2]), (double)2.0));
    }
}

//Wrapper for euclidean difference kernel
void euclideanDifferenceKernelWrapper(float *im_1, float *im_2, size_t noLibIm,
                                      unsigned char *mask_im,
                                      size_t size, size_t channels, size_t *target_area,
                                      double *variants, size_t blockSize)
{
    const size_t numBlocks = (size * size * noLibIm + blockSize - 1) / blockSize;
    euclideanDifferenceKernel<<<static_cast<unsigned int>(numBlocks),
                                static_cast<unsigned int>(blockSize)>>>(
        im_1, im_2, noLibIm, mask_im, size, channels, target_area, variants);
}

//Converts degrees to radians
__device__
constexpr double degToRadKernel(const double deg)
{
    return (deg * HIP_PI) / 180;
}

//Kernel that calculates the CIEDE2000 difference between main image and library images
__global__
void CIEDE2000DifferenceKernel(float *im_1, float *im_2, size_t noLibIm,
                               unsigned char *mask_im,
                               size_t size, size_t channels, size_t *target_area, double *variants)
{
    const size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * channels;
    const size_t stride = blockDim.x * gridDim.x * channels;
    for (size_t i = index; i < size * size * channels * noLibIm; i += stride)
    {
        const size_t im_1_index = i % (size * size * channels);
        const size_t grayscaleIndex = im_1_index / channels;

        const size_t row = grayscaleIndex / size;
        if (row < target_area[0] || row >= target_area[1])
        {
            variants[i / channels] = 0;
            continue;
        }

        const size_t col = grayscaleIndex % size;
        if (col < target_area[2] || col >= target_area[3])
        {
            variants[i / channels] = 0;
            continue;
        }

        if (mask_im[grayscaleIndex] == 0)
            variants[i / channels] = 0;
        else
        {
            const double k_L = 1.0, k_C = 1.0, k_H = 1.0;
            constexpr double deg360InRad = degToRadKernel(360.0);
            constexpr double deg180InRad = degToRadKernel(180.0);
            const double pow25To7 = 6103515625.0; //pow(25, 7)

            const double C1 = sqrt((double) (im_1[im_1_index + 1] * im_1[im_1_index + 1]) +
                    (im_1[im_1_index + 2] * im_1[im_1_index + 2]));
            const double C2 = sqrt((double) (im_2[i + 1] * im_2[i + 1]) +
                    (im_2[i + 2] * im_2[i + 2]));
            const double barC = (C1 + C2) / 2.0;

            const double G = 0.5 * (1 - sqrt(pow(barC, (double)7.0) / (pow(barC, (double)7.0) + pow25To7)));

            const double a1Prime = (1.0 + G) * im_1[im_1_index + 1];
            const double a2Prime = (1.0 + G) * im_2[i + 1];

            const double CPrime1 = sqrt((a1Prime * a1Prime) +
                                        (im_1[im_1_index + 2] * im_1[im_1_index + 2]));
            const double CPrime2 = sqrt((a2Prime * a2Prime) +(im_2[i + 2] * im_2[i + 2]));

            double hPrime1;
            if (im_1[im_1_index + 2] == 0 && a1Prime == 0.0)
                hPrime1 = 0.0;
            else
            {
                hPrime1 = atan2((double) im_1[im_1_index + 2], a1Prime);
                //This must be converted to a hue angle in degrees between 0 and 360 by
                //addition of 2 pi to negative hue angles.
                if (hPrime1 < 0)
                    hPrime1 += deg360InRad;
            }

            double hPrime2;
            if (im_2[i + 2] == 0 && a2Prime == 0.0)
                hPrime2 = 0.0;
            else
            {
                hPrime2 = atan2((double) im_2[i + 2], a2Prime);
                //This must be converted to a hue angle in degrees between 0 and 360 by
                //addition of 2pi to negative hue angles.
                if (hPrime2 < 0)
                    hPrime2 += deg360InRad;
            }

            const double deltaLPrime = im_2[i] - im_1[im_1_index];
            const double deltaCPrime = CPrime2 - CPrime1;

            double deltahPrime;
            const double CPrimeProduct = CPrime1 * CPrime2;
            if (CPrimeProduct == 0.0)
                deltahPrime = 0;
            else
            {
                //Avoid the fabs() call
                deltahPrime = hPrime2 - hPrime1;
                if (deltahPrime < -deg180InRad)
                    deltahPrime += deg360InRad;
                else if (deltahPrime > deg180InRad)
                    deltahPrime -= deg360InRad;
            }

            const double deltaHPrime = 2.0 * sqrt(CPrimeProduct) * sin(deltahPrime / 2.0);

            const double barLPrime = (im_1[im_1_index] + im_2[i]) / 2.0;
            const double barCPrime = (CPrime1 + CPrime2) / 2.0;

            double barhPrime;
            const double hPrimeSum = hPrime1 + hPrime2;
            if (CPrime1 * CPrime2 == 0.0)
                barhPrime = hPrimeSum;
            else
            {
                if (fabs(hPrime1 - hPrime2) <= deg180InRad)
                    barhPrime = hPrimeSum / 2.0;
                else
                {
                    if (hPrimeSum < deg360InRad)
                        barhPrime = (hPrimeSum + deg360InRad) / 2.0;
                    else
                        barhPrime = (hPrimeSum - deg360InRad) / 2.0;
                }
            }

            const double T = 1.0 - (0.17 * cos(barhPrime - degToRadKernel(30.0))) +
                    (0.24 * cos(2.0 * barhPrime)) +
                    (0.32 * cos((3.0 * barhPrime) + degToRadKernel(6.0))) -
                    (0.20 * cos((4.0 * barhPrime) - degToRadKernel(63.0)));

            const double deltaTheta = degToRadKernel(30.0) *
                    exp(-pow((barhPrime - degToRadKernel(275.0)) / degToRadKernel(25.0), 2.0));

            const double R_C = 2.0 * sqrt(pow(barCPrime, (double)7.0) /
                                          (pow(barCPrime, (double)7.0) + pow25To7));

            const double S_L = 1 + ((0.015 * pow(barLPrime - 50.0, (double)2.0)) /
                                    sqrt(20 + pow(barLPrime - 50.0, (double)2.0)));
            const double S_C = 1 + (0.045 * barCPrime);
            const double S_H = 1 + (0.015 * barCPrime * T);

            const double R_T = (-sin(2.0 * deltaTheta)) * R_C;


            variants[i / channels] = (double) sqrt(pow(deltaLPrime / (k_L * S_L), (double)2.0) +
                                                   pow(deltaCPrime / (k_C * S_C), (double)2.0) +
                                                   pow(deltaHPrime / (k_H * S_H), (double)2.0) +
                                                   (R_T * (deltaCPrime / (k_C * S_C)) *
                                                    (deltaHPrime / (k_H * S_H))));
        }
    }
}

//Wrapper for euclidean difference kernel
void CIEDE2000DifferenceKernelWrapper(float *im_1, float *im_2, size_t noLibIm,
                                      unsigned char *mask_im,
                                      size_t size, size_t channels, size_t *target_area,
                                      double *variants, size_t blockSize)
{
    const size_t numBlocks = (size * size * noLibIm + blockSize - 1) / blockSize;
    CIEDE2000DifferenceKernel<<<static_cast<unsigned int>(numBlocks),
                                static_cast<unsigned int>(blockSize)>>>(
        im_1, im_2, noLibIm, mask_im, size, channels, target_area, variants);
}

//Calculates repeats in range and adds to variants
__global__
void calculateRepeats(double *variants,
                     size_t *bestFit, const size_t bestFitMax, const size_t gridWidth,
                     const int leftRange, const int rightRange, const int upRange,
                     const size_t repeatAddition)
{
    for (int y = -upRange; y < 0; ++y)
    {
        for (int x = -leftRange; x <= rightRange; ++x)
        {
            if (bestFit[y * gridWidth + x] < bestFitMax)
                variants[bestFit[y * gridWidth + x]] += repeatAddition;
        }
    }
    for (int x = -leftRange; x < 0; ++x)
    {
        if (bestFit[x] < bestFitMax)
            variants[bestFit[x]] += repeatAddition;
    }
}

//Wrapper for calculate repeats kernel
void calculateRepeatsKernelWrapper(double *variants,
                                   size_t *bestFit, const size_t bestFitMax,
                                   const size_t gridWidth, const int x, const int y,
                                   const int padGrid,
                                   const size_t repeatRange, const size_t repeatAddition)
{
    const size_t paddedX = x + padGrid;
    const size_t paddedY = y + padGrid;

    const int leftRange = static_cast<int>(std::min(repeatRange, paddedX));
    const int rightRange = static_cast<int>(std::min(repeatRange, gridWidth - paddedX - 1));
    const int upRange = static_cast<int>(std::min(repeatRange, paddedY));
    calculateRepeats<<<1, 1>>>(variants,
                               bestFit + paddedY * gridWidth + paddedX, bestFitMax, gridWidth,
                               leftRange, rightRange, upRange,
                               repeatAddition);
}

//Finds lowest value in variants
__global__
void findLowestKernel(double *lowestVariant, size_t *bestFit, double *variants, size_t noLibIm)
{
    for (size_t i = 0; i < noLibIm; ++i)
    {
        if (variants[i] < *lowestVariant)
        {
            *lowestVariant = variants[i];
            *bestFit = i;
        }
    }
}

//Wrapper for find lowest kernel
void findLowestKernelWrapper(double *lowestVariant, size_t *bestFit, double *variants, size_t noLibIm)
{
    findLowestKernel<<<1, 1>>>(lowestVariant, bestFit, variants, noLibIm);
}
