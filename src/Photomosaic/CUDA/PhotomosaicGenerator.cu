#include "hip/hip_runtime.h"
/*
	Copyright © 2018-2020, Morgan Grundy

	This file is part of Mosaic Magnifique.

    Mosaic Magnifique is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Mosaic Magnifique is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Mosaic Magnifique.  If not, see <https://www.gnu.org/licenses/>.
*/
#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/hip_math_constants.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

#include "ColourDifference.cuh"

//Calculates the difference (using template function func) between two images (im_1, im_2) storing in variants
//Parts of the image can be ignored using im_mask (variant is set to 0)
//Image rows = size, cols = size
template<p_dfColourDifference func>
__global__
void imageDifference(float *im_1, float *im_2, unsigned char *im_mask,
                     const size_t size, double *variants)
{
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < size * size; i += stride)
    {
        variants[i] = func(im_1 + i * 3, im_2 + i * 3) * (im_mask[i] != 0);
    }
}

//Calculates the difference (using template function func) between two images (im_1, im_2) storing in variants
//Parts of the image can be ignored using im_mask (variant is set to 0)
//Image rows = size, cols = size, channels = channels
//Edge case equivalent of imageDifference:
//target_area contains bounds (min row, max row, min col, max col), variant set to 0 for out of bound pixels
template<p_dfColourDifference func>
__global__
void imageDifferenceEdge(float *im_1, float *im_2, unsigned char *mask_im,
                         const size_t size, size_t *target_area, double *variants)
{
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    const size_t imageSize = size * size;
    for (size_t i = index; i < imageSize; i += stride)
    {
        const size_t row = i / size;
        const size_t col = i % size;
        bool valid = row >= target_area[0];
        valid = row < target_area[1] && valid;
        valid = col >= target_area[2] && valid;
        valid = col < target_area[3] && valid;
        valid = mask_im[i] != 0 && valid;
        variants[i] = func(im_1 + i*3, im_2 + i*3) * valid;
    }
}

//Explicit instantiation of templates
template __global__ void imageDifference<euclideanDifference>(float *im_1, float *im_2, unsigned char *im_mask, const size_t size, double *variants);
template __global__ void imageDifference<CIEDE2000Difference>(float *im_1, float *im_2, unsigned char *im_mask, const size_t size, double *variants);
template __global__ void imageDifferenceEdge<euclideanDifference>(float *im_1, float *im_2, unsigned char *im_mask, const size_t size, size_t *target_area, double *variants);
template __global__ void imageDifferenceEdge<CIEDE2000Difference>(float *im_1, float *im_2, unsigned char *im_mask, const size_t size, size_t *target_area, double *variants);

//Wrapper for imageDifference kernel with euclideanDifference kernel
//target_area is unused, it is just there so the function parameters match the edge case one
void euclideanDifferenceKernelWrapper(float *main_im, float *lib_im, unsigned char *mask_im,
                                      const size_t size, [[maybe_unused]]size_t *target_area, double *variants,
                                      const size_t blockSize, hipStream_t stream)
{
    const size_t numBlocks = (size * size + blockSize - 1) / blockSize;
    imageDifference<euclideanDifference><<<static_cast<unsigned int>(numBlocks),
                               static_cast<unsigned int>(blockSize),
                               0, stream>>>(main_im, lib_im, mask_im, size, variants);
}

//Wrapper for imageDifference kernel with euclideanDifference kernel
void euclideanDifferenceEdgeKernelWrapper(float *main_im, float *lib_im, unsigned char *mask_im,
                                          const size_t size, size_t *target_area, double *variants,
                                          const size_t blockSize, hipStream_t stream)
{
    const size_t numBlocks = (size * size + blockSize - 1) / blockSize;
    imageDifferenceEdge<euclideanDifference><<<static_cast<unsigned int>(numBlocks),
                                  static_cast<unsigned int>(blockSize),
                                  0, stream>>>(main_im, lib_im, mask_im, size, target_area, variants);
}

//Wrapper for imageDifference kernel with CIEDE2000Difference kernel
//target_area is unused, it is just there so the function parameters match the edge case one
void CIEDE2000DifferenceKernelWrapper(float *main_im, float *lib_im, unsigned char *mask_im,
                                      const size_t size, [[maybe_unused]]size_t *target_area, double *variants,
                                      const size_t blockSize, hipStream_t stream)
{
    const size_t numBlocks = (size * size + blockSize - 1) / blockSize;
    imageDifference<CIEDE2000Difference><<<static_cast<unsigned int>(numBlocks),
                               static_cast<unsigned int>(blockSize),
                               0, stream>>>(main_im, lib_im, mask_im, size, variants);
}

//Wrapper for imageDifference kernel with CIEDE2000Difference kernel
void CIEDE2000DifferenceEdgeKernelWrapper(float *main_im, float *lib_im, unsigned char *mask_im,
                                          const size_t size, size_t *target_area, double *variants,
                                          const size_t blockSize, hipStream_t stream)
{
    const size_t numBlocks = (size * size + blockSize - 1) / blockSize;
    imageDifferenceEdge<CIEDE2000Difference><<<static_cast<unsigned int>(numBlocks),
                                   static_cast<unsigned int>(blockSize),
                                   0, stream>>>(main_im, lib_im, mask_im, size, target_area, variants);
}

//Calculates repeats in range and adds to variants
__global__
void calculateRepeats(double **variants, size_t noMainImage,
                     size_t *bestFit, const size_t bestFitMax, const size_t gridWidth,
                     const int leftRange, const int rightRange, const int upRange,
                     const size_t repeatAddition)
{
    for (int y = -upRange; y < 0; ++y)
    {
        for (int x = -leftRange; x <= rightRange; ++x)
        {
            if (bestFit[y * gridWidth + x] < bestFitMax)
            {
                for (size_t mainI = 0; mainI < noMainImage; ++mainI)
                    variants[mainI][bestFit[y * gridWidth + x]] += repeatAddition;
            }
        }
    }
    for (int x = -leftRange; x < 0; ++x)
    {
        if (bestFit[x] < bestFitMax)
        {
            for (size_t mainI = 0; mainI < noMainImage; ++mainI)
                variants[mainI][bestFit[x]] += repeatAddition;
        }
    }
}

//Wrapper for calculate repeats kernel
//Calculates repeats in range and adds to variants
void calculateRepeatsKernelWrapper(double **variants, size_t noMainImage,
                                   size_t *bestFit, const size_t bestFitMax,
                                   const size_t gridWidth, const int x, const int y,
                                   const int padGrid,
                                   const size_t repeatRange, const size_t repeatAddition)
{
    const size_t paddedX = x + padGrid;
    const size_t paddedY = y + padGrid;

    const int leftRange = static_cast<int>(std::min(repeatRange, paddedX));
    const int rightRange = static_cast<int>(std::min(repeatRange, gridWidth - paddedX - 1));
    const int upRange = static_cast<int>(std::min(repeatRange, paddedY));
    calculateRepeats<<<1, 1>>>(variants, noMainImage,
                               bestFit + paddedY * gridWidth + paddedX, bestFitMax, gridWidth,
                               leftRange, rightRange, upRange,
                               repeatAddition);
}

//Finds lowest value in variants
__global__
void findLowestKernel(double *lowestVariant, size_t *bestFit, double **variants, const size_t noLibIm, const size_t noMainImage)
{
    for (size_t mainI = 0; mainI < noMainImage; ++mainI)
    {
        for (size_t libI = 0; libI < noLibIm; ++libI)
        {
            if (variants[mainI][libI] < *lowestVariant)
            {
                *lowestVariant = variants[mainI][libI];
                *bestFit = libI;
            }
        }
    }
}

//Wrapper for find lowest kernel
//Finds lowest value in variants
void findLowestKernelWrapper(double *lowestVariant, size_t *bestFit, double **variants, const size_t noLibIm, const size_t noMainImage)
{
    findLowestKernel<<<1, 1>>>(lowestVariant, bestFit, variants, noLibIm, noMainImage);
}

//Flattens size elements that have spacing elements between them
//Grid x size should be set to size of the first dim of data
//Block x size should be set to at most spacing
__global__
void flattenKernel(double **data, const size_t size, const size_t spacing)
{
    for (size_t i = threadIdx.x; i < size; i += blockDim.x)
    {
        data[blockIdx.x][i] = data[blockIdx.x][i * spacing];
        __syncthreads();
    }
}

//Wrapper for flatten kernel
//Flattens size elements that have spacing elements between them
//Grid x size should be set to size of the first dim of data
//Block x size should be set to at most spacing
void flattenKernelWrapper(double **data, const size_t noMainImage, const size_t size, const size_t spacing, const size_t maxBlockSize)
{
    //Only need to flatten if spacing is > 1
    if (spacing > 1)
    {
        const size_t blockSize = std::min(maxBlockSize, spacing);
        flattenKernel<<<static_cast<unsigned int>(noMainImage),
            static_cast<unsigned int>(blockSize)>>>(data, size, spacing);
    }
}